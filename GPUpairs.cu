#include <stdio.h>
#include <stdlib.h>
#include <stdbool.h>
#include <hip/hip_runtime.h>

int main() {
    int devCount;
    hipGetDeviceCount(&devCount);
    printf("No. Devices: %d\n", devCount);
    return 0;
}

